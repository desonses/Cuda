/*
autor fredy m
uaem
desonses@gmail.com para mas comentarios
*/


#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_fp16.h"


/*
En el siguiente ejemplo se muestran las diferencias y las similitudes que existen a
la hora de reservar memoria tanto en el host como en el device. En este ejemplo se
reserva espacio para una matriz cuadrada de NxN elementos, se inicializa
en el host con valores aleatorios (entre 0 y 9) de tipo float y despues se transfieren los datos
desde el host hasta el device:
*/


#define N 8
// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{
	// declaracion
	float *hstA_matriz;
	float *devA_matriz;
	float *hstB_matriz;////
	float *devB_matriz;////

	// reserva en el host
	hstA_matriz = (float*)malloc(N*N * sizeof(float));
	hstB_matriz = (float*)malloc(N*N * sizeof(float));/////
	// reserva en el device
	hipMalloc((void**)&devA_matriz, N*N * sizeof(float));
	hipMalloc((void**)&devB_matriz, N*N * sizeof(float));////
	// inicializacion de datos
	srand((int)time(NULL));
	for (int i = 0; i < N*N; i++)
	{
		hstA_matriz[i] = (float)(rand() % 2);
	}

	// copia de datos
	hipMemcpy(devA_matriz, hstA_matriz, N*N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devB_matriz, devA_matriz, N*N * sizeof(float), hipMemcpyDeviceToDevice);
	hipMemcpy(hstB_matriz, devB_matriz, N*N * sizeof(float), hipMemcpyDeviceToHost);


	// salida
	printf("matriz A\n");
	for (int j = 0; j < N*N; j++)
	{

		printf("%f, ",hstA_matriz[j]);
	}
	printf("\n\n");
	printf("matriz B\n");
	for (int k = 0; k < N*N; k++)
	{

		printf("%f, ", hstB_matriz[k]);
	}

	hipFree(devA_matriz);
	hipFree(devB_matriz);
	printf("\npulsa INTRO para finalizar...");
	fflush(stdin);
	char tecla = getchar();
	return 0;
}


