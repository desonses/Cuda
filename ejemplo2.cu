#include "hip/hip_runtime.h"
/*
autor fredy m
uaem
desonses@gmail.com para mas comentarios
*/


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include ""

#define N 16 //tamano de los vectores
#define BLOCKS 5 // tamano del bloque(numero de hilos en cada bloque)


gridDim.x: La primera nos da el n�mero de bloques (M)
blockDim.x: la segunda el n�mero de hilos que tiene cada bloque (N)

/*
En este este ejemplo se realiza un ejemplo sencillo de suma de vectores (entrada x entrada)
*/

//Global: funcion llamada desde el host y ejecutada en el device(kernel)
__global__ void Add(float *a, float *b, float *c)
{
	int Id = threadIdx.x + blockDim.x * blockIdx.x;
	printf("(%d, %d, %d) ", threadIdx.x, blockDim.x, blockIdx.x);
	printf("hilo: %d, ", Id);
	//solo trabajan los N hilos
	if (Id < N) {
		c[Id] = a[Id] * b[Id];
	}
}



int main(int argc, char **argv)
{
	float *vector1, *vector2, *resultado;
	float *dev_vector1, *dev_vector2, *dev_resultado;

	//reserva de memoria en el host
	vector1 = (float*)malloc(N * sizeof(float));
	vector2 = (float*)malloc(N * sizeof(float));
	resultado = (float*)malloc(N * sizeof(float));

	//reserva de memoria en el device
	hipMalloc((void**)&dev_vector1, N * sizeof(float));
	hipMalloc((void**)&dev_vector2, N * sizeof(float));
	hipMalloc((void**)&dev_resultado, N * sizeof(float));

	// inicializacion de los vectores
	for (int i = 0; i < N; i++) {
		vector1[i] = (float) rand() / RAND_MAX;
		vector2[i] = (float) rand() / RAND_MAX;
	}

	//enviar los datos hacia el Device
	hipMemcpy(dev_vector1, vector1, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_vector2, vector2, N * sizeof(float), hipMemcpyHostToDevice);

	//lanzamiento del kernel
	//calculamos el numero de bloques necesario para un tamano de bloque fijo 
	int nBloques = N / BLOCKS;
	if (N % BLOCKS != 0) {
		nBloques = nBloques + 1;

	}


	int hilosB = BLOCKS;
	printf("\n");
	printf("vector de %d elementos\n", N);
	printf("Lanzamiento con %d bloques (%d hilos)\n", nBloques, nBloques*hilosB);

	Add <<<nBloques,hilosB>>>(dev_vector1, dev_vector2, dev_resultado);

	//recogida de los datos

	hipMemcpy(resultado, dev_resultado, N*sizeof(float),hipMemcpyDeviceToHost);

	//impresion de los datos
	printf(">vector1: \n");
	for (int i = 0; i < N;i++) {
		printf("%.2f, ", vector1[i]);

	}
	printf("\n");
	printf(">vector2: \n");
	for (int i = 0; i < N; i++) {
		printf("%.2f, ", vector2[i]);

	}
	printf("\n");
	printf(">suma: \n");
	for (int i = 0; i < N; i++) {
		printf("%.2f, ", resultado[i]);

	}

	printf("\n");

	//liberacion de memoria del device y host

	hipFree(dev_vector1);
	hipFree(dev_vector2);
	hipFree(dev_resultado);
	free(vector1);
	free(vector2);
	free(resultado);
	printf("\n...");
	fflush(stdin);
	char tecla = getchar();

	return 0;

}



