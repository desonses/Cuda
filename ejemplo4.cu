/*
autor fredy m
uaem
desonses@gmail.com para mas comentarios
*/



#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define N 24 //tamano de los vectores
#define BLOCKS 6 // tamano del bloque(numero de hilos en cada bloque)


/*
suma de vectores de 3 dim, mide el tiempo de ejecucion
*/


// gridDim.x: La primera nos da el n�mero de bloques (M)
// blockDim.x: la segunda el n�mero de hilos que tiene cada bloque (N)



//Global: funcion llamada desde el host y ejecutada en el device(kernel)

__global__ void Add(float *a, float *b, float *c)
{
	int Id = threadIdx.x + blockDim.x * blockIdx.x;
	if (Id < N) {
		a[Id] = threadIdx.x;
		b[Id] = blockIdx.x;
		c[Id] = Id;
	}
}



// funcion para revision de errores en las funciones de CUDA

__host__ void check_CUDA_Error(const char *mensaje) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), mensaje);	
	}
}



int main(int argc, char **argv)
{
	float *resultado1, *resultado2, *resultado3;
	float *resultado11, *resultado12, *resultado13;
	float *resultado21, *resultado22, *resultado23;

	float *dev_vector1, *dev_vector2, *dev_vector3;
	float *dev_vector11, *dev_vector12, *dev_vector13;
	float *dev_vector21, *dev_vector22, *dev_vector23;

	//reserva de memoria en el host
	resultado1 = (float*)malloc(N * sizeof(float));
	resultado2 = (float*)malloc(N * sizeof(float));
	resultado3 = (float*)malloc(N * sizeof(float));
	//reserva de memoria en el host
	resultado11 = (float*)malloc(N * sizeof(float));
	resultado12 = (float*)malloc(N * sizeof(float));
	resultado13 = (float*)malloc(N * sizeof(float));
	//reserva de memoria en el host
	resultado21 = (float*)malloc(N * sizeof(float));
	resultado22 = (float*)malloc(N * sizeof(float));
	resultado23 = (float*)malloc(N * sizeof(float));

	hipError_t error;
	//reserva de memoria en el device
	error = hipMalloc((void**)&dev_vector1, N * sizeof(float));
	error = hipMalloc((void**)&dev_vector2, N * sizeof(float));
	error = hipMalloc((void**)&dev_vector3, N * sizeof(float));

	error = hipMalloc((void**)&dev_vector11, N * sizeof(float));
	error = hipMalloc((void**)&dev_vector12, N * sizeof(float));
	error = hipMalloc((void**)&dev_vector13, N * sizeof(float));

	error = hipMalloc((void**)&dev_vector21, N * sizeof(float));
	error = hipMalloc((void**)&dev_vector22, N * sizeof(float));
	error = hipMalloc((void**)&dev_vector23, N * sizeof(float));

	if (error != hipSuccess) {
		printf("\n ocurrio un error: %s", hipGetErrorString(error));
	}

	//lanzamiento del kernel
	//calculamos el numero de bloques necesario para un tamano de bloque fijo 
	int nBloques = N / BLOCKS;
	if (N % BLOCKS != 0) {
		nBloques = nBloques + 1;
	}

	int nBloques2 = 1;
	int hilosB = BLOCKS;
	int hilosB2 = 1;


	// declaracion de eventos para medir el tiempo de ejecucion en la GPU
	hipEvent_t start;
	hipEvent_t stop;
	// creacion de eventos
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// marca de inicio
	hipEventRecord(start, 0);

	// codigo a temporizar en el device
	//ejecucion del kernel
	Add <<<nBloques, hilosB>>>(dev_vector1, dev_vector2, dev_vector3);
	check_CUDA_Error("Error Kernel 1");
	Add <<<nBloques2, N>>>(dev_vector11, dev_vector12, dev_vector13);
	check_CUDA_Error("Error Kernel 2");
	Add <<<N, 1>>>(dev_vector21, dev_vector22, dev_vector23);
	check_CUDA_Error("Error Kernel 3");
	// marca de final
	hipEventRecord(stop, 0);
	// sincronizacion GPU-CPU
	hipEventSynchronize(stop);
	// calculo del tiempo en milisegundos
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	

	//recogida de los datos
	hipMemcpy(resultado1, dev_vector1, N * sizeof(float), hipMemcpyDeviceToHost);
	check_CUDA_Error("Error dev_vector1");
	hipMemcpy(resultado2, dev_vector2, N * sizeof(float), hipMemcpyDeviceToHost);
	check_CUDA_Error("Error dev_vector2");
	hipMemcpy(resultado3, dev_vector3, N * sizeof(float), hipMemcpyDeviceToHost);
	check_CUDA_Error("Error dev_vector3");
	hipMemcpy(resultado11, dev_vector11, N * sizeof(float), hipMemcpyDeviceToHost);
	check_CUDA_Error("Error dev_vector11");
	hipMemcpy(resultado12, dev_vector12, N * sizeof(float), hipMemcpyDeviceToHost);
	check_CUDA_Error("Error dev_vector12");
	hipMemcpy(resultado13, dev_vector13, N * sizeof(float), hipMemcpyDeviceToHost);
	check_CUDA_Error("Error dev_vector13");
	hipMemcpy(resultado21, dev_vector21, N * sizeof(float), hipMemcpyDeviceToHost);
	check_CUDA_Error("Error dev_vector21");
	hipMemcpy(resultado22, dev_vector22, N * sizeof(float), hipMemcpyDeviceToHost);
	check_CUDA_Error("Error dev_vector22");
	hipMemcpy(resultado23, dev_vector23, N * sizeof(float), hipMemcpyDeviceToHost);
	check_CUDA_Error("Error dev_vector23");

	//impresion de los datos
	printf("\n");
	printf("vector de %d elementos\n", N);
	printf("Lanzamiento con %d bloques y %d hilos en cada bloque (%d hilos)\n", nBloques, BLOCKS,nBloques*hilosB);
	printf(">indice de hilo: \n");
	for (int i = 0; i < N; i++) {
		printf("%.0f, ", resultado1[i]);
	}
	printf("\n");
	printf(">indice de bloque: \n");
	for (int i = 0; i < N; i++) {
		printf("%.0f, ", resultado2[i]);
	}
	printf("\n");
	printf(">indice global: \n");
	for (int i = 0; i < N; i++) {
		printf("%.0f, ", resultado3[i]);
	}
	
	//
	printf("\n");
	printf("\n");
	printf("vector de %d elementos\n", N);
	printf("Lanzamiento con %d bloques (%d hilos)\n", nBloques2, nBloques*hilosB);

	//impresion de los datos
	printf(">indice de hilo: \n");
	for (int i = 0; i < N; i++) {
		printf("%.0f, ", resultado11[i]);
	}
	printf("\n");
	printf(">indice de bloque: \n");
	for (int i = 0; i < N; i++) {
		printf("%.0f, ", resultado12[i]);
	}
	printf("\n");
	printf(">indice global: \n");
	for (int i = 0; i < N; i++) {
		printf("%.0f, ", resultado13[i]);
	}

	//
	printf("\n");
	printf("\n");
	printf("vector de %d elementos\n", N);
	printf("Lanzamiento con %d bloques (%d hilos)\n", N, hilosB2);

	//impresion de los datos
	printf(">indice de hilo: \n");
	for (int i = 0; i < N; i++) {
		printf("%.0f, ", resultado21[i]);
	}
	printf("\n");
	printf(">indice de bloque: \n");
	for (int i = 0; i < N; i++) {
		printf("%.0f, ", resultado22[i]);
	}
	printf("\n");
	printf(">indice global: \n");
	for (int i = 0; i < N; i++) {
		printf("%.0f, ", resultado23[i]);
	}

	printf("\n");
	printf("\n");

	// impresion de resultados
	printf("> Tiempo de ejecucion: %f ms\n", elapsedTime);
	
	//liberacion de memoria del device
	hipFree(dev_vector1);
	hipFree(dev_vector2);
	hipFree(dev_vector3);
	hipFree(dev_vector11);
	hipFree(dev_vector12);
	hipFree(dev_vector13);
	hipFree(dev_vector21);
	hipFree(dev_vector22);
	hipFree(dev_vector23);

	//liberacion de memoria del host
	free(resultado1);
	free(resultado2);
	free(resultado3);
	free(resultado11);
	free(resultado12);
	free(resultado13);
	free(resultado21);
	free(resultado22);
	free(resultado23);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("\n...");
	fflush(stdin);
	char tecla = getchar();
	
	return 0;

}






