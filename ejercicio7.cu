/*
autor fredy m
uaem
desonses@gmail.com para mas comentarios
*/


#ifdef __HIPCC__
#define cuda_SYNCTHREADS() __syncthreads();
#else
#define cuda_SYNCTHREADS()
#endif

#include <hip/device_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <math.h>



/*
sincronizacion de hilos, verificacion de posibles errores, 
suma las potencias de elementos de un vector en el orden de log2(n)
*/


#define N 8
__device__ float valores(float, float);

__host__ void check_CUDA_Error(const char *mensaje)
{
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), mensaje);
	}
}


__global__ void reduccion(float *vector, float *suma)
{
	//reserva de memoria en la zona de memoria compartida
	__shared__ float temporal[N];

	//indice local de cada hilo -> kernel con un solo bloque
	int Id = threadIdx.x;

	//copiamos en 'temporal' el vector y sincronizamos los hilos
	temporal[Id] = vector[Id];
	
	cuda_SYNCTHREADS();
	//reduccion paralela
	int salto = N / 2;

	//realizamos log2(N) iteraciones
	while (salto)
	{
		//solo trabajan la mitad de los hilos
		if (Id < salto)
		{	

			temporal[Id] = (1 / powf(temporal[Id], 2)) + (1 / powf(temporal[Id + salto], 2));
			printf("temporal: %.3f\n", temporal[Id]);
		}
		//cuda_SYNCTHREADS();
		cuda_SYNCTHREADS();
		salto = salto / 2;
	}
	//el hilo 0 escribe el resultado final en la  memoria global
	if (Id == 0)
	{
		*suma = temporal[Id];
	}
}

__device__ float valores(float valor1, float valor2) {

	float suma = (1 / pow(valor1, 2)) + (1 / pow(valor2, 2));
	return suma;
}

int main(int argc, char** argv) 
{
	float *vector1, *resultado;
	float *dev_vector1, *dev_resultado;
	int size = N * sizeof(float);
	//reserva de memoria en el host
	vector1 = (float*)malloc(size);
	resultado = (float*)malloc(size);

	//reserva de memoria en el device
	hipMalloc((void**)&dev_vector1, size);
	hipMalloc((void**)&dev_resultado, size);

	// inicializacion de los vectores
	for (int i = 0; i < N; i++) {
		vector1[i] = (float)i + 1;
	}

	//enviar los datos hacia el Device
	hipMemcpy(dev_vector1, vector1, size, hipMemcpyHostToDevice);

	//lanzamiento del kernel

	reduccion<<<1, N>>>(dev_vector1, dev_resultado);

	//recogida de los datos

	hipMemcpy(resultado, dev_resultado, size, hipMemcpyDeviceToHost);

	//impresion de los datos
	printf("\n>vector1: \n");
	for (int i = 0; i < N; i++) {
		printf("%.3f, ", 1/pow(vector1[i],2));

	}

	printf("\n");
	printf(">suma: \n");
	for (int i = 0; i < N; i++) {
		printf("%.3f, ", resultado[i]);

	}

	printf("\n");

	//liberacion de memoria del device y host

	hipFree(dev_vector1);
	hipFree(dev_resultado);
	free(vector1);
	free(resultado);
	printf("\n...");
	fflush(stdin);
	char tecla = getchar();

	return 0;

}



