/*
autor fredy m
uaem
desonses@gmail.com para mas comentarios
*/



#include <hip/device_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <math.h>



#define N 33

/*
realiza la transpuesta de una matriz
*/

// definicio de memoria constante CUDA
__constant__ float dev_A[N][N];

//GLOBAL: func desde el host y ejecutada en el kernel(DEVICE)

__global__ void transpuesta(float *dev_B) 
{
	int columna = threadIdx.x;
	int fila = threadIdx.y;
	int pos = columna + N * fila;
	
	// cada hilo coloca un elemento de la matriz final
	dev_B[pos] = dev_A[columna][fila];
}


int main(int argc, char** argv)
{
	float *hst_A, *hst_B;
	float *dev_B;
	int size = N * N * sizeof(float);

	//reserva de memoria en el host
	hst_A = (float*)malloc(size);
	hst_B = (float*)malloc(size);

	//reserva de memoria en el device
	hipMalloc((void**)&dev_B, size);

	//llenar la matriz

	for (int i = 0; i < N*N; i++)
	{
		hst_A[i] = float(i) + 1;

	}

	//copiar los datos hacia el device
	hipError_t error = hipMemcpyToSymbol(HIP_SYMBOL(dev_A), hst_A, size);
	if (error != hipSuccess) {
		printf("Error Memori const\n");
	}

	//dimensiones del kernel a lanzar
	dim3 bloques(1);
	dim3 hilos(N, N);

	//lanzamiento del kernel
	transpuesta <<<bloques, hilos >>> (dev_B);

	//recoger los datos
	hipMemcpy(hst_B, dev_B, size, hipMemcpyDeviceToHost);

	//impresion de los datos
	printf("Matriz original:\n");
	
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++)
		{
			printf("%.2f\t", hst_A[j + i * N]);
		}
		printf("\n");
	}
	
	printf("Matriz transpuesta:\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%.2f\t", hst_B[j + i * N]);
		}
		printf("\n");
	}
	
	//
	printf("\n pulsa INTRO para salir:\n");
	fflush(stdin);
	char tecla = getchar();

	return 0;
}
