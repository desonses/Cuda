/*
autor fredy m
uaem
desonses@gmail.com para mas comentarios
*/



#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>


#define N 20

/*
operaciones con matrices
*/

//GLOABL: funcion llamada desde el host y ejecutada en el device (kernel)

__global__ void suma(float *A, float *B, float *C)
{
	//indice de las columnas
	int columna = threadIdx.x;
	//indice de las filas
	int fila = threadIdx.y;
	//indice lineal
	int Id = columna + fila * blockDim.x;
	//sumamos cada elemento
	C[Id] = A[Id] + B[Id];
}


//Cada elemento de la matriz C se obtiene como la suma de los
//elementos de la matriz A ubicados en posiciones adyacentes

__global__ void add(float *A, float *C) 
{
	int columna = threadIdx.x;
	//indice de las filas
	int fila = threadIdx.y;
	//indice lineal
	int Id = columna + fila * blockDim.x;

	int id1 = (columna - 1) + fila * blockDim.x;
	int id2 = (columna + 1) + fila * blockDim.x;
	int id3 = columna + (fila - 1) * blockDim.x;
	int id4 = columna + (fila + 1) * blockDim.x;

	if ((fila > 0 && fila < N - 1) && (columna > 0 && columna < N - 1)) {

		C[Id] = A[id1] + A[id2] + A[id3] + A[id4];
	}
	else
	{
		C[Id] = A[Id];
	}
}


// funcion chequeo de errores

__host__ void check_CUDA_Error(const char *mensaje) 
{
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), mensaje);
	}
}



int main(int argc, char** argv) 
{
	//declaracions
	float *hst_A, *hst_B, *hst_C;
	float *dev_A, *dev_B, *dev_C;
	int size = N * N * sizeof(float);

	//reserva de memoria en el host
	hst_A = (float*)malloc(size);
	hst_B = (float*)malloc(size);
	hst_C = (float*)malloc(size);

	//reserva de memoria en el device
	hipMalloc((void**)&dev_A, size);
	check_CUDA_Error("Error malloc dev_A!");
	hipMalloc((void**)&dev_B, size);
	check_CUDA_Error("Error malloc dev_B!");
	hipMalloc((void**)&dev_C, size);
	check_CUDA_Error("Error malloc dev_C!");

	//inicializacion de los vectores

	for (int i = 0; i < N*N; i++) {

		hst_A[i] = (float)(rand() % 5);
		hst_B[i] = (float)(rand() % 5);
	}

	//enviar datos del hosto al device
	
	hipMemcpy(dev_A, hst_A, size, hipMemcpyHostToDevice);
	check_CUDA_Error("Error Memcpy hst_A To dev_A");
	hipMemcpy(dev_B, hst_B, size, hipMemcpyHostToDevice);
	check_CUDA_Error("Error Memcpy hst_B To dev_B");
	//cudaMemcpy(dev_C, hst_C, N*N * sizeof(float), cudaMemcpyHostToDevice);
	hipMemcpy(dev_C, hst_C, size, hipMemcpyHostToDevice);
	check_CUDA_Error("Error Memcpy hst_C To dev_C");
	//dimenciones del kernel
	dim3 Nbloques(1);
	dim3 hilosB(N, N);

	//////MEDICION DE TIEMPO EN GPU///////////////
	// declaracion de eventos para medir el tiempo de ejecucion en la GPU
	hipEvent_t start;
	hipEvent_t stop;
	// creacion de eventos
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// marca de inicio
	hipEventRecord(start, 0);
	//llamada al kernel dibimensional de NxN hilos
	//suma <<<Nbloques, hilosB >>> (dev_A, dev_B, dev_C);
	add<<<Nbloques, hilosB >>>(dev_A, dev_C);
	check_CUDA_Error("Error kernel");
	
	// marca de final
	hipEventRecord(stop, 0);
	// sincronizacion GPU-CPU
	hipEventSynchronize(stop);
	// calculo del tiempo en milisegundos
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	////////MEDICION DE TIEMPO EN GPU/////////////////

	//recodiga de los datos
	hipMemcpy(hst_C, dev_C, size, hipMemcpyDeviceToHost);
	check_CUDA_Error("Error Memcpy dev_C To hst_C");
	//impresion del resultado
	
	printf("A:\n");
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++)
		{
			printf("%.2f\t", hst_A[j + i * N]);
		}
		printf("\n");
	}

	
	printf("B:\n");
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++)
		{
			printf("%.2f\t", hst_B[j + i * N]);
		}
		printf("\n");
	}
	
	printf("C:\n");
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++)
		{
			printf("%.2f\t", hst_C[j + i * N]);
		}
		printf("\n");
	}
	printf("\n\n");
	printf("> Tiempo de ejecucion: %f ms\n", elapsedTime);

	hipFree(hst_A);
	hipFree(hst_B);
	hipFree(hst_C);
	free(hst_A);
	free(hst_B);
	free(hst_C);

	printf("\n pulsa INTRO parsa finalizar...");
	fflush(stdin);
	char tecla = getchar();

	return 0;
}

