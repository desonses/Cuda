/*
autor fredy m
uaem
desonses@gmail.com para mas comentarios
*/



#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


/*
ejemplo que llena un vector que invierte sus valores
*/

#define N 30 //tamano de los vectores

__global__ void invierte(float *a, float *b) {
	int id = threadIdx.x;
	//int id = threadIdx.x + blockDim.x * blockIdx.x;// para n-bloques de 1 hilo

	if (id < N) 
	{
		b[id] = a[N-id];
	}
}



__host__ void check_CUDA_Error(const char *mensaje) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), mensaje);
	}
}


int main(int argc, char** argv)
{	
	float *vector1, *resultado;
	float *dev_vector1, *dev_resultado;

	//reserva de memoria en el host
	vector1 = (float*)malloc(N * sizeof(float));
	resultado = (float*)malloc(N * sizeof(float));

	//reserva de memoria en el device
	hipMalloc((void**)&dev_vector1, N * sizeof(float));
	check_CUDA_Error("Error Malloc dev_vector");
	hipMalloc((void**)&dev_resultado, N * sizeof(float));
	check_CUDA_Error("Error Malloc dev_resultado");

	// inicializacion de los vectores
	printf("vector inicial: \n");
	for (int i = 0; i < N; i++) {
		vector1[i] = (float)rand() / RAND_MAX;
		printf("%.2f, ", vector1[i]);
	}
	
	//enviar los datos hacia el Device
	hipMemcpy(dev_vector1, vector1, N * sizeof(float), hipMemcpyHostToDevice);
	check_CUDA_Error("Error CudaMemcpy");
	
	//MEDICION DE TIEMPO EN GPU
	// declaracion de eventos para medir el tiempo de ejecucion en la GPU
	hipEvent_t start;
	hipEvent_t stop;
	// creacion de eventos
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// marca de inicio
	hipEventRecord(start, 0);
	
	//Add <<<nBloques,hilosB>>>(dev_vector1, dev_vector2, dev_resultado);
	invierte<<<1, N >>>(dev_vector1, dev_resultado);
	// cambiar (N,1) para n bloques de 1 hilo
	check_CUDA_Error("Error Kernel");
	
	// marca de final
	hipEventRecord(stop, 0);
	// sincronizacion GPU-CPU
	hipEventSynchronize(stop);
	// calculo del tiempo en milisegundos
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	//MEDICION DE TIEMPO EN GPU
	
	//recogida de los datos
	printf("\n");
	printf("vector de regreso:\n");
	hipMemcpy(resultado, dev_resultado, N * sizeof(float), hipMemcpyDeviceToHost);
	check_CUDA_Error("Error CudaMemcpy2");
	for (int i = 0; i < N; i++) {
		printf("%.2f, ", resultado[i]);

	}
	// impresion de resultados
	printf("\n");
	printf("> Tiempo de ejecucion: %f ms\n", elapsedTime);

	return 0;

	hipFree(dev_vector1);
	hipFree(dev_resultado);
	free(vector1);
	free(resultado);

}





