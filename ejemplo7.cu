/*
autor fredy m
uaem
desonses@gmail.com para mas comentarios
*/


#ifdef __HIPCC__
#define cuda_SYNCTHREADS() __syncthreads()
#else
#define cuda_SYNCTHREADS()
#endif

#include <hip/device_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>

#include <hip/hip_runtime_api.h>


#define N 16
/*
suma de los elementos de un vector en el orden de log2(n)
*/

//kernel

__global__ void reduccion(float *vector, float *suma)
{
	//reserva de memoria en la zona de memoria compartida
	__shared__ float temporal[N];

	//indice local de cada hilo -> kernel con un solo bloque
	int Id = threadIdx.x;

	//copiamos en 'temporal' el vector y sincronizamos los hilos
	temporal[Id] = vector[Id];
	cuda_SYNCTHREADS();
	
	//reduccion paralela
	int salto = N / 2;

	//realizamos log2(N) iteraciones
	while (salto)
	{
		//solo trabajan la mitad de los hilos
		if (Id < salto)
		{
			temporal[Id] = temporal[Id] + temporal[Id + salto];
		}
		//cuda_SYNCTHREADS();
		cuda_SYNCTHREADS();
		salto = salto / 2;
	}
	//el hilo 0 escribe el resultado final en la  memoria global
	if (Id == 0)
	{
		*suma = temporal[Id];
	}
}


int main(int argc, char** argv)
{

	float *vector1,  *resultado;
	float *dev_vector1, *dev_resultado;
	int size = N * sizeof(float);
	//reserva de memoria en el host
	vector1 = (float*)malloc(size);
	resultado = (float*)malloc(size);

	//reserva de memoria en el device
	hipMalloc((void**)&dev_vector1, size);
	hipMalloc((void**)&dev_resultado, size);

	// inicializacion de los vectores
	for (int i = 0; i < N; i++) {
		vector1[i] = (float)rand() / RAND_MAX;

	}

	//enviar los datos hacia el Device
	hipMemcpy(dev_vector1, vector1, size, hipMemcpyHostToDevice);

	//lanzamiento del kernel
	
	reduccion <<<1, N >>> (dev_vector1, dev_resultado);

	//recogida de los datos

	hipMemcpy(resultado, dev_resultado, size, hipMemcpyDeviceToHost);

	//impresion de los datos
	printf(">vector1: \n");
	for (int i = 0; i < N; i++) {
		printf("%.3f, ", vector1[i]);

	}

	printf("\n");
	printf(">suma: \n");
	for (int i = 0; i < N; i++) {
		printf("%.3f, ", resultado[i]);

	}

	printf("\n");

	//liberacion de memoria del device y host

	hipFree(dev_vector1);
	hipFree(dev_resultado);
	free(vector1);
	free(resultado);
	printf("\n...");
	fflush(stdin);
	char tecla = getchar();

	return 0;

}
