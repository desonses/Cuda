#include "hip/hip_runtime.h"
/*
autor fredy m
uaem
desonses@gmail.com para mas comentarios
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>
#include "cpu_bitmap.h"

// Defines
#define DIM 1024 // Dimensiones del Bitmap

/*
generacion de una imagen en RGBa
*/

// GLOBAL: funcion llamada desde el host y ejecutada en el device (kernel)

__global__ void kernel(unsigned char *imagen)
{
	// coordenada horizontal
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	// coordenada vertical
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	// coordenada global de cada pixel
	int pixel = x + y * blockDim.x * gridDim.x;
	
	// cada hilo pinta un pixel con un color arbitrario

	imagen[pixel * 4 + 0] = 255 * x / (blockDim.x * gridDim.x/8); // canal R
	imagen[pixel * 4 + 1] = 255 * y / (blockDim.y * gridDim.y/8); // canal G
	imagen[pixel * 4 + 2] = 2 * blockIdx.x + 2 * blockIdx.y/8; // canal B
	imagen[pixel * 4 + 3] = 255; // canal alfa

}



// MAIN: rutina principal ejecutada en el host

int main(int argc, char** argv)
{
	// declaracion del bitmap
	CPUBitmap bitmap(DIM, DIM);

	// tama�o en bytes
	size_t size = bitmap.image_size();
	
	// reserva en el host
	unsigned char *host_bitmap = bitmap.get_ptr();
	
	// reserva en el device
	unsigned char *dev_bitmap;
	hipMalloc((void**)&dev_bitmap, size);
	
	// generamos el bitmap
	dim3 Nbloques(DIM / 16, DIM / 16);
	dim3 hilosB(16, 16);
	kernel <<<Nbloques, hilosB >>> (dev_bitmap);
	
	// recogemos el bitmap desde la GPU para visualizarlo
	hipMemcpy(host_bitmap, dev_bitmap, size, hipMemcpyDeviceToHost);
	
	// liberacion de recursos
	hipFree(dev_bitmap);
	
	// visualizacion y salida
	printf("\n...pulsa ESC para finalizar...");
	bitmap.display_and_exit();
	return 0;
}


