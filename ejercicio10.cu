/*
autor fredy m
uaem
desonses@gmail.com para mas comentarios
*/

#ifdef __HIPCC__
#define cuda_SYNCTHREADS() __syncthreads();
#else
#define cuda_SYNCTHREADS()
#endif

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <math.h>

#define N 32

/*
calcula el valor aproximado de pi, realizando particiones (entre mas, hay mas aproximacion al valor)
*/

__host__ float func(float valor) 
{
	return  4 / (1 + powf(valor,2));
}

__global__ void calcula(float *particion, float *funcion, float *sum)
{
	//reserva dinamica de memoria compartida en tiempo de ejecucion
	extern __shared__ float temporal[];
	float add[N];
	//float h = (1 - 0) / N;
	int id = threadIdx.x;// +blockIdx.x * blockDim.x;
	
	float xi, xim; 
	float yi, yim;

	//printf("%.2f, \n", particion[id]);
	xi = particion[id];
	xim = particion[id - 1];
	yi = funcion[id];
	yim = funcion[id - 1];
	add[id] = .5f * ((xi - xim) * (yi + yim));
	temporal[id] = add[id];

	printf("(%.4f - %.4f) * (%.4f + %.4f): %.4f\n", xi, xim, yi, yim, temporal[id]);
	
	cuda_SYNCTHREADS();
	//reduccion paralela
	int salto = N / 2;
	
	//realizamos log2(N) iteraciones
	while (salto)
	{
		//solo trabajan la mitad de los hilos
		if (id < salto)
		{
			temporal[id] = temporal[id] + temporal[id + salto];
		}
		//cuda_SYNCTHREADS();
		cuda_SYNCTHREADS();
		salto = salto / 2;
	}
	//el hilo 0 escribe el resultado final en la  memoria global
	if (id == 0)
	{
		*sum = temporal[id];
		//printf("temporal: %.3f\n", *sum);
	}


}

int main(int argc, char** argv)
{
	float *vector1, *vector2, *resultado;
	float *dev_vector1, *dev_vector2, *dev_resultado;
	size_t size = N * sizeof(float);

	//reserva de memoria en el host
	vector1 = (float*)malloc(size);
	vector2 = (float*)malloc(size);
	resultado = (float*)malloc(size);

	//reserva de memoria en el device
	hipMalloc((void**)&dev_vector1, size);
	hipMalloc((void**)&dev_vector2, size);
	hipMalloc((void**)&dev_resultado, size);

	// inicializacion de los vectores
	for (int i = 0; i < N; i++) {
		vector1[i] = (float)i / (N - 1);
		vector2[i] = func(vector1[i]);
		//printf("xi: %.2f, f(xi): %.2f \n", vector1[i], vector2[i]);
	}


	//enviar los datos hacia el Device
	hipMemcpy(dev_vector1, vector1, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_vector2, vector2, size, hipMemcpyHostToDevice);
	//cudaMemcpy(dev_resultado, resultado, size, cudaMemcpyHostToDevice);

	//lanzamiento del kernel con memoria dinamica compartida
	calcula <<<1, N, size>>>(dev_vector1, dev_vector2, dev_resultado);

	//recogida de los datos

	hipMemcpy(resultado, dev_resultado, size, hipMemcpyDeviceToHost);
	printf("pi = %.5f, \n", resultado[0]);

	return 0;
}
