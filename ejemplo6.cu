/*
autor fredy m
uaem
desonses@gmail.com para mas comentarios
*/



#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <>

/*
muestra los hilos disponibles por grid en tu targeta cuda
*/

// estructura de dim3
// dim3 blocks(Bx, By, Bz);
// dim3 threads(hx, hy, hz);

gridDim.x = Bx
gridDim.y = By
gridDim.z = Bz
blockDim.x = hx
blockDim.y = hy
blockDim.z = hz



int main(int argc, char** argv)
{
	hipDeviceProp_t deviceProp;
	int deviceID;
	hipGetDevice(&deviceID);
	hipGetDeviceProperties(&deviceProp, deviceID);
	printf("MAX threads per block: %d\n", deviceProp.maxThreadsPerBlock);
	printf("MAX BLOCK SIZE\n");
	printf(" [x -> %d]\n [y -> %d]\n [z -> %d]\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
	printf("MAX GRID SIZE\n");
	printf(" [x -> %d]\n [y -> %d]\n [z -> %d]\n"), deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2];
	

	//dim3 bloques(3, 2, 1);
	//dim3 hilos(16, 16, 1);
	
	printf("\n pulsa INTRO parsa finalizar...");
	fflush(stdin);
	char tecla = getchar();


	return 0;


}

