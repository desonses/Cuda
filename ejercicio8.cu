/*
autor fredy m
uaem
desonses@gmail.com para mas comentarios
*/

#include <hip/device_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define N 3
/*
multiplicacion de matrices con memoria constante
*/

// definicion de memoria constante CUDA

__constant__ float dev_A[N][N];
__constant__ float dev_B[N][N];


//GLOBAL: func desde el host y ejecutada en el kernel(DEVICE)


__global__ void multiplicacion(float *dev_C)
{
	int suma = 0;
	int columna = threadIdx.x;
	int fila = threadIdx.y;
	int pos = columna + N * fila;

	if (columna < N && fila < N) 
	{
		for (int k = 0; k < N; k++) 
		{
			dev_C[pos] += dev_A[fila][k] * dev_B[k][columna];
		}
	}
}



int main(int argc, char** argv)
{
	float *hst_A, *hst_B, *hst_C;
	float *dev_C;
	int size = N * N * sizeof(float);

	//reserva de memoria en el host
	hst_A = (float*)malloc(size);
	hst_B = (float*)malloc(size);
	hst_C = (float*)malloc(size);
	//reserva de memoria en el device
	hipMalloc((void**)&dev_C, size);

	//llenar la matriz
	for (int i = 0; i < N*N; i++)
	{
		hst_A[i] = float(i) + 1;
		hst_B[i] = float(i);
	}

	//copiar los datos hacia el device desde memoria constante
	hipError_t error = hipMemcpyToSymbol(HIP_SYMBOL(dev_A), hst_A, size);
	if (error != hipSuccess) {
		printf("Error Memoria constante dev_A to hst_A\n");
	}
	error = hipMemcpyToSymbol(HIP_SYMBOL(dev_B), hst_B, size);
	if (error != hipSuccess) {
		printf("Error Memoria constante dev_B to hst_B\n");
	}

	//dimensiones del kernel a lanzar
	dim3 bloques(1);
	dim3 hilos(N, N);

	//lanzamiento del kernel
	multiplicacion <<<bloques, hilos >>> (dev_C);

	//recoger los datos
	hipMemcpy(hst_C, dev_C, size, hipMemcpyDeviceToHost);

	//impresion de los datos
	printf("\nMatriz A:\n");

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++)
		{
			printf("%.2f\t", hst_A[j + i * N]);
		}
		printf("\n");
	}
	printf("\nMatriz B:\n");
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++)
		{
			printf("%.2f\t", hst_B[j + i * N]);
		}
		printf("\n");
	}
	printf("\n");
	printf("multiplicacion de matrices A y B:\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%.2f\t", hst_C[j + i * N]);
		}
		printf("\n");
	}
	
	printf("\n pulsa INTRO para salir:\n");
	fflush(stdin);
	char tecla = getchar();

	return 0;
}

