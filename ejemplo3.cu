
/*
autor fredy m
uaem
desonses@gmail.com para mas comentarios
*/



#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_fp16.h>

/*
En este ejercicio se implemente un prog para resolver ecuaciones de segundo grado. 

*/

// Device: kernel que se ejecuta en la GPU
__global__ void suma_GPU(int a, int b, int *c)
{
	*c = a + b;
}

// solve equation second grade
__global__ void solve_GPU(int a, int b, int c ,int *x1, int *x2)
{
	int raiz = powf(b, 2) - (4 * a * c);
	int i = -b / 2 * a;
	int j = 2 * a;

	*x1 = i + sqrtf(raiz) / j;
	*x2 = i - sqrtf(raiz) / j;
}


// HOST: funcion llamada y ejecutada desde el host
__host__ int suma_CPU(int a, int b)
{
	return (a + b);
}


int main(int argc, char** argv)
{

	// declaraciones
	int n1 = 1, n2 = 2, c = 0;
	int *hst_c;

	int *hst_x1;
	int *hst_x2;

	int m1 = 10, m2 = 20;
	int *dev_c;

	// equacion
	int a = 1, b =8 , C = -6;
	int *dev_x1;
	int *dev_x2;

	// reserva de memoria en el host
	//hst_c = (int*)malloc( sizeof(int) );
	
	hst_x1 = (int*)malloc(sizeof(int));
	hst_x2 = (int*)malloc(sizeof(int));


	// reserva de memoria en el device
	//cudaMalloc((void**)&dev_c, sizeof(int) );

	hipMalloc((void**)&dev_x1, sizeof(int));
	hipMalloc((void**)&dev_x2, sizeof(int));



	// llamada a la funcion suma_CPU
	//c = suma_CPU(n1, n2);

	// resultados CPU
	//printf("CPU:\n");
	//printf("%2d + %2d = %2d \n",n1, n2, c);

	// llamada a la funcion suma_GPU
	//suma_GPU<<<1,1>>>(m1, m2, dev_c);

	solve_GPU<<<1,1>>>(a,b,C, dev_x1, dev_x2);

	// recogida de datos desde el device hacia el host
	//cudaMemcpy(hst_c, dev_c, sizeof(int), cudaMemcpyDeviceToHost );

	hipMemcpy(hst_x1, dev_x1, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(hst_x2, dev_x2, sizeof(int), hipMemcpyDeviceToHost);

	printf("resultado: \n");
	printf("x1 = %2d ,x2 = %2d \n", *hst_x1, *hst_x2);

	// resultados GPU
	//printf("GPU:\n");
	//printf("%2d + %2d = %2d \n",m1, m2, *hst_c);

	// salida
	printf("\npulsa INTRO para finalizar...");
	fflush(stdin);
	char tecla = getchar();

	free(hst_c);//liberacion de memoria del host
	hipFree(dev_c);//liberacion de memoria del device(kernel)
	return 0;

}


